#include "hip/hip_runtime.h"
﻿// Program to calculate vectors with CUDA and comparing speed difference between CPU and GPU.

#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#include <hip/hip_runtime.h>
#include <>

// Function for GPU calculation
__global__ void vector_addition(int* a, int* b, int* c) {
	int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
	c[tid] = a[tid] + b[tid];
}


// Function for filling the arrays with numbers between 0-32767
void fill_array(int* data) {
	for (int i = 0; i < 32768; i++) {
		data[i] = i;
	}
}

// CPU calculation and result print function
void calculate_print(int* a, int* b, int* c) {
	printf("#### CPU ####\n");
	for (int i = 0; i < 32768; i++) {
		c[i] = a[i] + b[i];
		printf("%d + %d = %d\n", a[i], b[i], c[i]);
	}
	printf("------ CPU COMPLETE ------ \n Press 'ENTER' to continue...\n");
}

// GPU result print function
void print_array(int* a, int* b, int* c) {
	printf("\n#### GPU ####\n");
	for (int i = 0; i < 32768; i++) {
		printf("%d + %d = %d\n", a[i], b[i], c[i]);
	}
	printf("------ GPU COMPLETE ------");
}

int main() {

	clock_t t_gpu, t_cpu;

	int a[32768], b[32768], c[32768];
	int h_a[32768], h_b[32768], h_c[32768];
	int* d_a, * d_b, * d_c;

	size_t size = sizeof(int) * 32768;

	fill_array(a);
	fill_array(b);

	fill_array(h_a);
	fill_array(h_b);

	hipMalloc(&d_a, size);
	hipMalloc(&d_b, size);
	hipMalloc(&d_c, size);

	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

	t_cpu = clock();
	calculate_print(h_a, h_b, h_c);
	t_cpu = clock() - t_cpu;
	double time_taken_cpu = ((double)t_cpu) / CLOCKS_PER_SEC;

	getchar();

	t_gpu = clock();
	vector_addition << <32768, 1 >> > (d_a, d_b, d_c);

	hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

	print_array(a, b, c);
	t_gpu = clock() - t_gpu;
	double time_taken_gpu = ((double)t_gpu) / CLOCKS_PER_SEC;

	printf("\n ##### STATISTICS #####");
	printf("\n - CPU took %lf seconds to complete.\n", time_taken_cpu);
	printf(" - GPU took %lf seconds to complete.\n", time_taken_gpu);

	if (time_taken_cpu > time_taken_gpu) {
		double speed = time_taken_cpu / time_taken_gpu;
		printf(" - GPU is %lf times faster than the CPU.", speed);
	}
	else {
		double speed = time_taken_gpu / time_taken_cpu;
		printf(" - CPU is %lf times faster than the GPU.", speed);
	}

	return 0;
}